#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#define MIN_VALUE (-1e38)

template <typename F>
__global__ void kernel_forward(const int B, const int T, const int C,
                               const F *__restrict__ const _w, const F *__restrict__ const _u, const F *__restrict__ const _k, const F *__restrict__ const _v,
                               F *__restrict__ const _y) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _c = idx % C;
    const int _offset = _b * T * C + _c;

    F u = _u[_c];
    F w = _w[_c];
    const F *__restrict__ const k = _k + _offset;
    const F *__restrict__ const v = _v + _offset;
    F *__restrict__ const y = _y + _offset;

    // aa and bb are running sums divided by exp(pp) (to avoid overflow)
    F aa = 0, bb = 0, pp = MIN_VALUE;
    for (int i = 0; i < T; i++) {
        const int ii = i * C;
        const F kk = k[ii];
        const F vv = v[ii];

        F ww = u + kk;
        F p = max(pp, ww);
        F e1 = exp(pp - p);
        F e2 = exp(ww - p);
        y[ii] = (e1 * aa + e2 * vv) / (e1 * bb + e2);
        
        ww = w + pp;
        p = max(ww, kk);
        e1 = exp(ww - p);
        e2 = exp(kk - p);
        aa = e1 * aa + e2 * vv;
        bb = e1 * bb + e2;
        pp = p;
    }
}

template <typename F>
__global__ void kernel_forward_with_state(
    const int B, const int T, const int C, const F *__restrict__ const _w, const F *__restrict__ const _u,
    const F *__restrict__ const _k, const F *__restrict__ const _v, F *__restrict__ const _y, F *__restrict__ const _s
) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _c = idx % C;
    const int _offset_s = _b * C * 3 + _c * 3;
    const int _offset = _b * T * C + _c;

    F u = _u[_c];
    F w = _w[_c];
    const F *__restrict__ const k = _k + _offset;
    const F *__restrict__ const v = _v + _offset;
    F *__restrict__ const y = _y + _offset;
    F *__restrict__ const s = _s + _offset_s;

    // aa and bb are running sums divided by exp(pp) (to avoid overflow)
    F aa = s[0], bb = s[1], pp = s[2];
    for (int i = 0; i < T; i++) {
        const int ii = i * C;
        const F kk = k[ii];
        const F vv = v[ii];

        F ww = u + kk;
        F p = max(pp, ww);
        F e1 = exp(pp - p);
        F e2 = exp(ww - p);
        y[ii] = (e1 * aa + e2 * vv) / (e1 * bb + e2);
        
        ww = w + pp;
        p = max(ww, kk);
        e1 = exp(ww - p);
        e2 = exp(kk - p);
        aa = e1 * aa + e2 * vv;
        bb = e1 * bb + e2;
        pp = p;
    }
    s[0] = aa;
    s[1] = bb;
    s[2] = pp;
}

template <typename F>
__global__ void kernel_backward(const int B, const int T, const int C,
                                const F *__restrict__ const _w, const F *__restrict__ const _u, const F *__restrict__ const _k, const F *__restrict__ const _v,
                                const F *__restrict__ const _y, const F *__restrict__ const _gy,
                                F *__restrict__ const _gw, F *__restrict__ const _gu, F *__restrict__ const _gk, F *__restrict__ const _gv) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _c = idx % C;
    const int _offset = _b * T * C + _c;

    F u = _u[_c];
    F w = _w[_c];
    const F *__restrict__ const k = _k + _offset;
    const F *__restrict__ const v = _v + _offset;
    const F *__restrict__ const y = _y + _offset;
    const F *__restrict__ const gy = _gy + _offset;
    F *__restrict__ const gk = _gk + _offset;
    F *__restrict__ const gv = _gv + _offset;

    F q[Tmax], r[Tmax];

    F gw = 0, gu = 0, aa = 0, bb = 0, ga = 0, gb = 0, pp = MIN_VALUE;
    for (int i = 0; i < T; i++) {
        const int ii = i * C;
        const F kk = k[ii];
        const F vv = v[ii];
        const F yy = y[ii];

        F ww = u + kk;
        F p = max(pp, ww);
        F e1 = exp(pp - p);
        F e2 = exp(ww - p);
        const F qq = gy[ii] / (e1 * bb + e2);
        gw += (ga - gb * yy) * e1 * qq;
        gu += (vv - yy) * e2 * qq;
        q[i] = qq;
        r[i] = ww - p;

        ww = w + pp;
        p = max(ww, kk);
        e1 = exp(ww - p);
        e2 = exp(kk - p);
        ga = e1 * (aa + ga);
        gb = e1 * (bb + gb);
        aa = e1 * aa + e2 * vv;
        bb = e1 * bb + e2;
        pp = p;
    }
    const int _offsetBC = _b * C + _c;
    _gw[_offsetBC] = gw * _w[_c]; // multiply by w because of w -> -exp(w) in python forward()
    _gu[_offsetBC] = gu;

    aa = 0, bb = 0, pp = MIN_VALUE;
    for (int i = T - 1; i >= 0; i--) {
        const int ii = i * C;
        const F kk = k[ii];
        const F vv = v[ii];
        const F yy = y[ii];
        const F qq = q[i];
        const F rr = r[i];

        F e1 = qq * exp(rr);
        F e2 = exp(kk + pp);
        gk[ii] = e1 * (vv - yy) + e2 * (aa * vv + bb);
        gv[ii] = e1 + e2 * aa;

        const F ww = w + pp;
        const F www = rr - u - kk;
        const F p = max(ww, www);
        e1 = exp(ww - p);
        e2 = qq * exp(www - p);
        aa = e1 * aa + e2;
        bb = e1 * bb - e2 * yy;
        pp = p;
    }
}

__global__ void kernel_state_forward(const int B, const int T, const int C,
                               const float *__restrict__ const _w, const float *__restrict__ const _u, const float *__restrict__ const _k, const float *__restrict__ const _v,
                               const float *__restrict__ const last_state, float *__restrict__ const _y, float *__restrict__ const new_state) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _c = idx % C;
    const int _offset = _b * T * C + _c;
    const int state_offset = (_b * C + _c)*3;

    float u = float(_u[_c]);
    float w = _w[_c];
    const float *__restrict__ const k = _k + _offset;
    const float *__restrict__ const v = _v + _offset;
    float *__restrict__ const y = _y + _offset;

    float p, q, o;
    if (last_state == NULL) {
        p = 0, q = 0, o = MIN_VALUE;
    } else {
        p = last_state[state_offset+0];
        q = last_state[state_offset+1];
        o = last_state[state_offset+2];
    }
    // p and q are running sums divided by exp(o) (to avoid overflows)
    for (int i = 0; i < T; i++) {
        const int ii = i * C;
        const float kk = float(k[ii]);
        const float vv = float(v[ii]);

        float no = max(o, u + kk);
        float A = exp(o - no);
        float B = exp(u + kk - no);
        y[ii] = float((A * p + B * vv) / (A * q + B));

        no = max(w + o, kk);
        A = exp(w + o - no);
        B = exp(kk - no);
        p = A * p + B * vv;
        q = A * q + B;
        o = no;
    }
    if (new_state != NULL) {
        new_state[state_offset+0] = p;
        new_state[state_offset+1] = q;
        new_state[state_offset+2] = o;
    }
}

__global__ void kernel_state_backward(const int B, const int T, const int C,
                                const float *__restrict__ const _w, const float *__restrict__ const _u, const float *__restrict__ const _k, const float *__restrict__ const _v, const float *__restrict__ const last_state, 
                                const float *__restrict__ const _gy, const float *__restrict__ const gnew_state,
                                float *__restrict__ const _gw, float *__restrict__ const _gu, float *__restrict__ const _gk, float *__restrict__ const _gv, float *__restrict__ const glast_state) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _c = idx % C;
    const int _offset = _b * T * C + _c;
    const int state_offset  = (_b * C + _c)*3;

    float u = float(_u[_c]);
    float w = _w[_c];
    const float *__restrict__ const k = _k + _offset;
    const float *__restrict__ const v = _v + _offset;
    const float *__restrict__ const gy = _gy + _offset;

    float *__restrict__ const gk = _gk + _offset;
    float *__restrict__ const gv = _gv + _offset;

    float y[Tmax], z[Tmax], zexp[Tmax];

    float gw = 0, gu = 0;
    float dpdw = 0, dqdw = 0;
    float p, q, o;
    if (last_state == NULL) {
        p = 0, q = 0, o = MIN_VALUE;
    } else {
        p = last_state[state_offset+0];
        q = last_state[state_offset+1];
        o = last_state[state_offset+2];
    }
    for (int i = 0; i < T; i++) {
        const int ii = i * C;
        const float kk = float(k[ii]);
        const float vv = float(v[ii]);
        const float gyy = float(gy[ii]);

        float no = max(o, kk + u);
        float A = exp(o - no);
        float B = exp(kk + u - no);

        float num = A * p + B * vv;
        float iden = 1 / (A * q + B);

        y[i] = num * iden;
        z[i] = iden;
        zexp[i] = kk + u - no;

        gw += gyy * (dpdw - dqdw * y[i]) * iden * A;
        gu += gyy * (vv - y[i]) * B * iden;

        no = max(w + o, kk);
        A = exp(w + o - no);
        B = exp(kk - no);
        dpdw = A * (p + dpdw);
        dqdw = A * (q + dqdw);
        p = A * p + B * vv;
        q = A * q + B;
        o = no;
    }

    float gp = 0, gq = 0, go = MIN_VALUE;
    if (gnew_state != NULL) {
        gp = gnew_state[state_offset+0];
        gq = gnew_state[state_offset+1];
        go = gnew_state[state_offset+2];
        if (gp == 0 && gq == 0) go = MIN_VALUE;
        gw += (gp * dpdw + gq * dqdw) * exp(o+go);
    }

    for (int i = T - 1; i >= 0; i--) {
        const int ii = i * C;
        const float kk = float(k[ii]);
        const float vv = float(v[ii]);
        const float gyy = float(gy[ii]);

        float A = gyy * z[i] * exp(zexp[i]);
        float B = exp(kk + go);
        gk[ii] = float(A * (vv - y[i]) + B * (gp * vv + gq));
        gv[ii] = float(A + B * gp);

        float no = max(w + go, zexp[i] - kk - u);
        A = exp(w + go - no);
        B = gyy * z[i] * exp(zexp[i] - kk - u - no);
        gp = A * gp + B;
        gq = A * gq - B * y[i];
        go = no;
    }

    // glast_state[2] is not the gradient w.r.t of last_state[2]
    // o (index 2) in last_state is just an exponent for p and q
    // so there are really only 2 elements to differentiate on
    // Similary go (glast_state index 2) is just an exponent for gp and gq
    if (glast_state != NULL) {
        glast_state[state_offset+0] = gp;
        glast_state[state_offset+1] = gq;
        glast_state[state_offset+2] = go;
    }

    // Multiply by w because the w -> -exp(w) preprocessing is halfway in the backwards pass, even though it's not in the forward pass
    const int _offsetBC = _b * C + _c;
    _gw[_offsetBC] = float(gw * _w[_c]);
    _gu[_offsetBC] = gu;
}


void cuda_forward(int B, int T, int C, float *w, float *u, float *k, float *v, float *y) {
    dim3 threadsPerBlock( min(C, 32) ); // requires --maxrregcount 60 for optimal performance
    assert(B * C % threadsPerBlock.x == 0);
    dim3 numBlocks(B * C / threadsPerBlock.x);
    kernel_forward<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, y);
}

void cuda_forward_with_state(int B, int T, int C, float *w, float *u, float *k, float *v, float *y, float *s) {
    dim3 threadsPerBlock( min(C, 32) ); // requires --maxrregcount 60 for optimal performance
    assert(B * C % threadsPerBlock.x == 0);
    dim3 numBlocks(B * C / threadsPerBlock.x);
    kernel_forward_with_state<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, y, s);
}


void cuda_backward(int B, int T, int C, float *w, float *u, float *k, float *v, float *y, float *gy, float *gw, float *gu, float *gk, float *gv) {
    dim3 threadsPerBlock( min(C, 32) ); // requires --maxrregcount 60 for optimal performance
    assert(B * C % threadsPerBlock.x == 0);
    dim3 numBlocks(B * C / threadsPerBlock.x);
    kernel_backward<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, y, gy, gw, gu, gk, gv);
}

void cuda_state_forward(int B, int T, int C, float *w, float *u, float *k, float *v, float *last_state, float *y, float *new_state) {
    dim3 threadsPerBlock( min(C, 32) ); // requires --maxrregcount 60 for optimal performance
    assert(B * C % threadsPerBlock.x == 0);
    dim3 numBlocks(B * C / threadsPerBlock.x);
    kernel_state_forward<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, last_state, y, new_state);
}

void cuda_state_backward(int B, int T, int C, float *w, float *u, float *k, float *v, float *last_state, float *gy, float *gnew_state, float *gw, float *gu, float *gk, float *gv, float *glast_state) {
    dim3 threadsPerBlock( min(C, 32) ); // requires --maxrregcount 60 for optimal performance
    assert(B * C % threadsPerBlock.x == 0);
    dim3 numBlocks(B * C / threadsPerBlock.x);
    kernel_state_backward<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, last_state, gy, gnew_state, gw, gu, gk, gv, glast_state);
}
